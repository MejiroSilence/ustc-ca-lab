#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <fstream>
#include "include.hpp"
#include <string>

#define blockN 8

using namespace std;

int N_;
int N;
int matrixSize;

__global__ void blocked_gemm_baseline(float *A, float *B, float *C, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < (N - N % blockN) && y < (N - N % blockN))
    {
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int blockNum = N / blockDim.x;
        __shared__ float subA[blockN][blockN];
        __shared__ float subB[blockN][blockN];
        int aStart = blockIdx.x * blockN * N;
        int bStart = blockIdx.y * blockN;
        int aStep = blockN;
        int bStep = blockN * N;
        float temp = 0;
        for (int k = 0; k < blockNum; ++k)
        {
            subA[tx][ty] = A[aStart + k * aStep + tx * N + ty];
            subB[tx][ty] = B[bStart + k * bStep + tx * N + ty];
            __syncthreads();
            for (int i = 0; i < blockDim.x; ++i)
                temp += subA[tx][i] * subB[i][ty];
            __syncthreads();
        }
        C[x * N + y] = temp;
    }
    else if (x >= N || y >= N)
        return;
    else
    {
        float temp = 0;
        auto aptr = A + x * N;
        auto bptr = B + y;
        for (int i = 0; i < N; ++i)
        {
            // float += A[x * N + i] * B[i * N + y];
            temp += (*aptr) * (*bptr);
            aptr += 1;
            bptr += N;
        }
        C[x * N + y] = temp;
    }
}

void cpu_gemm_baseline(float *A, float *B, float *C)
{
    float *now = C;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            float sum = 0;
            auto a = A + i * N;
            auto b = B + j;
            for (int k = 0; k < N; ++k)
            {
                sum += (*a) * (*b);
                a += 1;
                b += N;
            }
            *now = sum;
            now += 1;
        }
    }
}

bool gemm_verify(float *A, float *B, float *C)
{
    auto base_c = new float[matrixSize];
    cpu_gemm_baseline(A, B, base_c);
    auto end = C + matrixSize;
    // ofstream baseout("base");
    // ofstream avxout("avx");
    // printMatrix(C, N, avxout);
    // printMatrix(base_c, N, baseout);

    for (float *p1 = C, *p2 = base_c; p1 != end; ++p1, ++p2)
    {
        if (*p1 != *p2)

            return 0;
    }
    /*
        for (int i = 0; i < N; ++i)
            for (int j = 0; j < N; ++j)
                if (C[i * N + j] != base_c[i * N + j])
                {
                    cout << '(' << i << ',' << j << ')' << endl;
                }*/

    return 1;
};

int main(int argc, char **argv)
{
    N_ = stoi(argv[1]);
    N = 1 << N_;
    matrixSize = N * N;
    auto a = new float[matrixSize];
    auto b = new float[matrixSize];
    auto c = new float[matrixSize];
    randInit(a, matrixSize, 10);
    randInit(b, matrixSize, 10);
    float *d_a, *d_b, *d_c;
    auto nBytes = matrixSize * sizeof(float);
    hipMalloc((void **)&d_a, nBytes);
    hipMalloc((void **)&d_b, nBytes);
    hipMalloc((void **)&d_c, nBytes);
    hipMemcpy((void *)d_a, (void *)a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_b, (void *)b, nBytes, hipMemcpyHostToDevice);
    dim3 blockSize(blockN, blockN);
    dim3 gridSize((N + blockN - 1) / blockN, (N + blockN - 1) / blockN);
    blocked_gemm_baseline<<<gridSize, blockSize, 2 * blockN * blockN * sizeof(float)>>>(d_a, d_b, d_c, N);
    hipMemcpy((void *)c, (void *)d_c, nBytes, hipMemcpyDeviceToHost);

    cout << (gemm_verify(a, b, c) ? "true" : "false");
}
